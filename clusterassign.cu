#include "hip/hip_runtime.h"

__global__ void clusterAssign(float *Cx, float *Cy, float *Cz, float *Px, float *Py, float *Pz, int *assigns, int Pwidth, int k) 
{
   int tid, i, temp_d, centerIdx;
   int d = MAX_INT;

   tid = blockIdx.x * blockDim.x + threadIdx.x;

   while (tid < width) {
      for (i = 0; i < k; i++) {
         temp_d = (Px[tid] - Cx[i])*(Px[tid] - Cx[i]) 
                    + (Py[tid] - Cy[i])*(Py[tid] - Cy[i]) 
                    + (Pz[tid] - Cz[i])*(Pz[tid] - Cz[i]);
         if (temp_d < d) {
            d = temp_d;
            centerIdx = i;
         }
      }
      assigns[tid] = centerIdx;
      tid += blockDim.x * gridDim.x;      
   }
   
   return;
}

extern "C" void cudaAssign(float *Cxin, float *Cyin, float *Czin, float *Pxin, float *Pyin, float *Pzin, int *assignments, int Pwidth, int k) {
   float *Cx, *Cy, *Cz, *Px, *Py, *Pz;
   int *assigns;

   hipMalloc(&Cx, (unsigned long)k*sizeof(float));
   hipMalloc(&Cy, (unsigned long)k*sizeof(float));
   hipMalloc(&Cz, (unsigned long)k*sizeof(float));
   hipMalloc(&Px, (unsigned long)Pwidth*sizeof(float));
   hipMalloc(&Py, (unsigned long)Pwidth*sizeof(float));
   hipMalloc(&Pz, (unsigned long)Pwidth*sizeof(float));
   hipMalloc(&assigns, (unsigned long)Pwidth*sizeof(int));

   hipMemcpy(Cx, Cxin, k*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Cy, Cyin, k*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Cz, Czin, k*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Px, Pxin, Pwidth*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Py, Pyin, Pwidth*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Pz, Pzin, Pwidth*sizeof(float), hipMemcpyHostToDevice);
   
   clusterAssign<<<1024, 512>>>(Cx, Cy, Cz, Px, Py, Pz, assigns, Pwidth, k);
   hipMemcpy(assignments, assigns, Pwidth*sizeof(int), hipMemcpyDeviceToHost);
   
   hipFree(Cx);
   hipFree(Cy);
   hipFree(Cz);   
   hipFree(Px);   
   hipFree(Py);   
   hipFree(Pz);   
   hipFree(assigns);   
}
