
#include <hip/hip_runtime.h>

__global__ void clusterAssign(float *Cx, float *Cy, float *Cz, float *Px, float *Py, float *Pz, int *assigns, int Pwidth, int k) 
{
   int tid, i, temp_d, centerIdx;
   int d;

   tid = blockIdx.x * blockDim.x + threadIdx.x;

   while (tid < Pwidth) {
      d = INT_MAX;
      for (i = 0; i < k; i++) {
         temp_d = (Px[tid] - Cx[i])*(Px[tid] - Cx[i]) 
                    + (Py[tid] - Cy[i])*(Py[tid] - Cy[i]) 
                    + (Pz[tid] - Cz[i])*(Pz[tid] - Cz[i]);
         if (temp_d < d) {
            d = temp_d;
            centerIdx = i;
         }
      }
      assigns[tid] = centerIdx;
      tid += blockDim.x * gridDim.x;      
   }
   
   return;
}

extern "C" void cudaAssign(float *Cxin, float *Cyin, float *Czin, float *Pxin, float *Pyin, float *Pzin, int *assignments, int Pwidth, int k) {
   float *Cx, *Cy, *Cz, *Px, *Py, *Pz;
   int *assigns;

   hipMalloc(&Cx, k*sizeof(float));
   hipMalloc(&Cy, k*sizeof(float));
   hipMalloc(&Cz, k*sizeof(float));
   hipMalloc(&Px, Pwidth*sizeof(float));
   hipMalloc(&Py, Pwidth*sizeof(float));
   hipMalloc(&Pz, Pwidth*sizeof(float));
   hipMalloc(&assigns, Pwidth*sizeof(int));

   hipMemcpy(Cx, Cxin, k*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Cy, Cyin, k*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Cz, Czin, k*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Px, Pxin, Pwidth*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Py, Pyin, Pwidth*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Pz, Pzin, Pwidth*sizeof(float), hipMemcpyHostToDevice);
   
   
   clusterAssign<<<1024, 512>>>(Cx, Cy, Cz, Px, Py, Pz, assigns, Pwidth, k);
   hipMemcpy(assignments, assigns, Pwidth*sizeof(int), hipMemcpyDeviceToHost);
   
   
   hipFree(Cx);
   hipFree(Cy);
   hipFree(Cz);   
   hipFree(Px);   
   hipFree(Py);   
   hipFree(Pz);   
   hipFree(assigns);   
}
